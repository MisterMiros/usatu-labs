#include "hip/hip_runtime.h"
#include "common.h"
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

size_t set_n(int argc, char** argv) {
    size_t N = MAX_N;
    if (argc == 2) {
        N = atoi(argv[1]);
    }
    if (argc == 3) {
        if (strcmp(argv[1],"-n") == 0) {
            N = atoi(argv[2]);
        }
        else if (strcmp(argv[1],"-mb") == 0) {
            N = (atoi(argv[2])*MB)/sizeof(int);
        }
    }
    return N > MAX_N ? MAX_N : N;
}

void random_ints(int* a, size_t size) {
    for (int i = 0; i<size; i++) {
        a[i] = rand() % 101 - 50;
    }
}

int check_results(int* a, int* b, int* c, size_t size) {
    int result = TRUE;
    for (int i = 0; i < size; i++) {
        if (c[i] != a[i]+b[i]) {
            result = FALSE;
        }
    }
    return result;
}

float add_calc_time(int* a,int* b,int* c,size_t N) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    add<<<(N+M-1)/M, M>>>(a, b, c, N); // Launch add() kernel on GPU with N blocks

    hipDeviceSynchronize();
    check_cuda_errors(__FILE__,__LINE__);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float calc_time = 0;
    hipEventElapsedTime(&calc_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return calc_time;
}

__global__ void add(int *a, int *b, int *c, size_t size) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) { c[index] = a[index] + b[index]; }
}

void check_cuda_errors(const char *filename, const int line_number) {
#ifdef DEBUG
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
        exit(-1);
    }
#endif
}
