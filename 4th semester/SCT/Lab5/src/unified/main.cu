#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <stdint.h>
#include "../include/common.h"

int main(int argc, char** argv) {
    size_t N = set_n(argc,argv);
    printf("N = %lu\n",N);
    srand(time(NULL));
    clock_t start_c = clock();
    int *a, *b, *c; // host copies of a, b, c
    size_t size = N * sizeof(int);
    // Alloc space for a, b, c
    hipMallocManaged((void **)&a, size);
    check_cuda_errors(__FILE__,__LINE__);
    random_ints(a, N);
    hipMallocManaged((void **)&b, size);
    check_cuda_errors(__FILE__,__LINE__);
    random_ints(b, N);
    hipMallocManaged((void **)&c, size);
    check_cuda_errors(__FILE__,__LINE__);

    float calc_time = add_calc_time(a,b,c,N);
    printf("GPU time: %.4f ms\n", calc_time);

    printf(check_results(a, b, c, N) ? "Result is correct\n" : "Result is wrong\n");
    // Cleanup
    hipFree(a); hipFree(b); hipFree(c);
    clock_t end_c = clock();
    printf("Total time: %.4f ms \n", ((double)(end_c-start_c)) * 1000 / CLOCKS_PER_SEC);
    return 0;
}
