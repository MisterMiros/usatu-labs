#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <stdint.h>
#include "../include/common.h"

int main(int argc, char** argv) {
    size_t N = set_n(argc,argv);
    printf("N = %lu\n",N);
    srand(time(NULL));
    clock_t start_c = clock();
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    size_t size = N * sizeof(int);
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    check_cuda_errors(__FILE__,__LINE__);
    hipMalloc((void **)&d_b, size);
    check_cuda_errors(__FILE__,__LINE__);
    hipMalloc((void **)&d_c, size);
    check_cuda_errors(__FILE__,__LINE__);
    //Alloc space for host copies of a, b, c
	//and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)calloc(N, sizeof(int));
    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    check_cuda_errors(__FILE__,__LINE__);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    check_cuda_errors(__FILE__,__LINE__);

    float calc_time = add_calc_time(d_a,d_b,d_c,N);
    printf("GPU time: %.4f ms\n", calc_time);

	// Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    check_cuda_errors(__FILE__,__LINE__);

    printf(check_results(a, b, c, N) ?
		"Result is correct\n" : "Result is wrong\n");
    // Cleanup
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    clock_t end_c = clock();
    printf("Total time: %.4f ms \n",
		((double)(end_c-start_c)) * 1000 / CLOCKS_PER_SEC);
    return 0;
}
