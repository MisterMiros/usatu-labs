#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <stdint.h>
#include "include/common.h"

#define CUDA_MALLOC(a,s) \
    hipMalloc((void **)&a,s); \
    check_cuda_errors(__FILE__,__LINE__);

#define CUDA_MALLOC_HOST(a,s) \
    hipHostMalloc((void**)&a,s); \
    check_cuda_errors(__FILE__,__LINE__);

int main(int argc, char** argv) {
    int nStreams = 32;
    size_t N = MAX_N;
    if (argc > 1) { nStreams = atoi(argv[1]); }
    if (argc > 2) {
        N = atoi(argv[2]);
        N = N > MAX_N ? MAX_N : N;
    }
    printf("N = %lu\n",N);
    printf("Stream Count = %d\n", nStreams);
    srand(time(NULL));
    clock_t start_c = clock();
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    size_t size = N * sizeof(int);
    // Alloc space for device copies of a, b, c
    CUDA_MALLOC(d_a,size);
    CUDA_MALLOC(d_b,size);
    CUDA_MALLOC(d_c,size);
    // Alloc space for host copies of a, b, c
    // and setup input values
    CUDA_MALLOC_HOST(a,size);
    random_ints(a, N);
    CUDA_MALLOC_HOST(b,size);
    random_ints(b, N);
    CUDA_MALLOC_HOST(c,size);

    hipStream_t* streams =
        (hipStream_t *)malloc(nStreams*sizeof(hipStream_t));
    for (int i = 0; i < nStreams; i++) {
        hipStreamCreate(&streams[i]);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start, 0);

    size_t width = N / nStreams;
    for (int i = 0; i < nStreams; i++) {
        size_t offset = i*width;
        hipMemcpyAsync(&d_a[offset], &a[offset],
            width*sizeof(int), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&d_b[offset], &b[offset],
            width*sizeof(int), hipMemcpyHostToDevice, streams[i]);
        add<<<(width+M-1)/M,M,0,streams[i]>>>
            (&d_a[offset],&d_b[offset],&d_c[offset], width);
    }
    for (int i = 0; i < nStreams; i++) {
        size_t offset = i*width;
        hipMemcpyAsync(&c[offset],&d_c[offset],
            width*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();
    check_cuda_errors(__FILE__,__LINE__);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float calc_time = 0;
    hipEventElapsedTime(&calc_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("GPU time: %.4f ms\n", calc_time);

    printf(check_results(a, b, c, N) ? "Result is correct\n" : "Result is wrong\n");
    // Cleanup
    for(int i = 0; i < nStreams; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipHostFree(a); hipHostFree(b); hipHostFree(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    clock_t end_c = clock();
    printf("Total time: %.4f ms \n", ((double)(end_c-start_c)) * 1000 / CLOCKS_PER_SEC);
    return 0;
}
