#include "hip/hip_runtime.h"
#include "common.h"
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

void random_ints(int* a, size_t size) {
    for (int i = 0; i<size; i++) {
        a[i] = rand() % 101 - 50;
    }
}

int check_results(int* a, int* b, int* c, size_t size) {
    int result = TRUE;
    for (int i = 0; i < size; i++) {
        if (c[i] != a[i]+b[i]) {
            //printf("%d, %d+%d=%d\n",i, a[i],b[i],c[i]);
            result = FALSE;
        }
    }
    return result;
}

__global__ void add(int *a, int *b, int *c, size_t size) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) { c[index] = a[index] + b[index]; }
}

void check_cuda_errors(const char *filename, const int line_number) {
#ifdef DEBUG
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
        exit(-1);
    }
#endif
}
